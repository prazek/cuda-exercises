#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrix_multiplication(const int *d_indices,
                                      const int *d_matrix,
                                      const int *d_vector,
                                      int *d_output,
                                      int n,
                                      int t) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n)
        return;

    int result = 0;
    for (int i = 0 ; i < t; i++) {
        const int multiplierIndex = index + d_indices[i];
        if (multiplierIndex < 0 || multiplierIndex > n)
            continue;
        const int elemIndex = n * i + index;
        result += d_matrix[elemIndex] * d_vector[multiplierIndex];
    }
    d_output[index] = result;
}

#define cudaCheckErrors(EXPR) assert(EXPR == hipSuccess)

/// Function for fast integer fetching
int fetch_int() {
    int result = 0;

    char c = 0;
    // skip all other chars
    while (c < '0' or c > '9') {
        c = getchar_unlocked();
    }

    while ('0' <= c and c <= '9') {
        result *= 10;
        result += c - '0';
        c = getchar_unlocked();
    }

    return result;
}


int main() {
    int n = fetch_int();
    int t = fetch_int();
    int *h_fullInput;

    // We need memory for n indices, n * t matrix elements and n elements of vector.
    const int full = n * (t + 2);
    cudaCheckErrors(hipHostMalloc((void**)&h_fullInput, sizeof(int) * full, hipHostMallocDefault));
    int *h_indices = h_fullInput;
    int *h_matrix = h_fullInput + n;
    int *h_vector = h_matrix + n * t;

    for (int i = 0; i < t; i++) {
        h_indices[i] = fetch_int();
        for (int j = 0 ; j < n ; j++) {
            int index = t * i + j;
            h_matrix[index] = fetch_int();
        }
    }

    for (int i = 0 ; i < n ; i++) {
        h_vector[i] = fetch_int();
    }

    int *d_fullInput;
    cudaCheckErrors(hipMalloc((void**)&d_fullInput, sizeof(int) * full));
    hipMemcpy(d_fullInput, h_fullInput, sizeof(int) * full, hipMemcpyHostToDevice);
    const int *d_indices = d_fullInput;
    const int *d_matrix = d_fullInput + n;
    const int *d_vector = d_matrix + n * t;

    int * d_output;
    cudaCheckErrors(hipMalloc((void**)&d_output, sizeof(int) * n));

    const int blockSize = 512;
    const int gridSize = (n + blockSize - 1) / blockSize;

    matrix_multiplication<<<gridSize, blockSize>>>(d_indices, d_matrix, d_vector, d_output, n, t);

    // write output into indices to save malloc call
    int * h_output = h_indices;
    hipMemcpy(h_output, d_output, sizeof(int) * n, hipMemcpyDeviceToHost);
    for (int i = 0 ; i < n ; i++) {
        printf("%d\n", d_output[i]);
    }

    hipFree(d_fullInput);
    hipFree(h_fullInput);
    hipFree(d_output);
}